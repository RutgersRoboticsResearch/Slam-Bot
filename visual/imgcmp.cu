#include "hip/hip_runtime.h"
__global__ void GPU_sum(float *G, float *F, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) {
    return;
  }
  double sum = F[i];
  for (int j = 1; (i % (j >> 1)) == 0 && (i + j) < n; j >>= 1) {
    sum += F[i + j];
    __syncthreads();
  }
  if (i == 0) {
    *G = sum;
  }
}

__global__ void GPU_sub(float *H, float *F, float *G, int n_rows, int n_cols) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n_rows || j >= n_cols) {
    return;
  }
  H[IJ2C(i, j, n_rows)] = F[IJ2C(i, j, n_rows)] - G[IJ2C(i, j, n_rows)];
}

__global__ void GPU_abs(float *H, float *F, int n_rows, int n_cols) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n_rows || j >= n_cols) {
    return;
  }
  int v = F[IJ2C(i, j, n_rows)];
  H[IJ2C(i, j, n_rows)] = v * ((v >= 0) - (v < 0));
}

__global__ void GPU_eemult(float *H, float *F, float *G, int n_rows, int n_cols) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n_rows || j >= n_cols) {
    return;
  }
  H[IJ2C(i, j, n_rows)] = F[IJ2C(i, j, n_rows)] * G[IJ2C(i, j, n_rows)];
}

__global__ void GPU_sad2(float *S, float *I1, float *I2, int n_rows, int n_cols) {

}

float sad2(const gcube &I1, const gcube &I2) {
  assert(I1.n_rows == I2.n_rows && I1.n_cols == I2.n_cols);
  float h_G;
  float *d_G;
  checkCudaErrors(hipMalloc(&d_G, sizeof(float)));
  gcube Temp(I1.n_rows, I1.n_cols);
  GPU_sub<<<(Temp.n_elem-1)/256+1, 256>>>(Temp, I1, I2);
  checkCudaErrors(hipGetLastError());
  GPU_abs<<<(Temp.n_elem-1)/256+1, 256>>>(Temp, Temp);
  checkCudaErrors(hipGetLastError());
  GPU_sum<<<(Temp.n_elem-1)/256+1, 256>>>(d_G, Temp);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(&h_G, d_G, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_G));
  return h_G;
}

float ssd2(const gcube &I1, const gcube &I2) {
  assert(I1.n_rows == I2.n_rows && I1.n_cols == I2.n_cols);
  float h_G;
  float *d_G;
  checkCudaErrors(hipMalloc(&d_G, sizeof(float)));
  gcube Temp(I1.n_rows, I1.n_cols);
  GPU_sub<<<(T.n_elem-1)/256+1, 256>>>(Temp, I1, I2);
  checkCudaErrors(hipGetLastError());
  GPU_eemult<<<(Temp.n_elem-1)/256+1, 256>>>(Temp, Temp, Temp);
  checkCudaErrors(hipGetLastError());
  GPU_sum<<<(Temp.n_elem-1)/256+1, 256>>>(d_G, Temp);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(&h_G, d_G, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_G));
  return h_G;
}

float ncc2(const gcube &I1, const gcube &I2) {
  assert(I1.n_rows == I2.n_rows && I1.n_cols == I2.n_cols);
  float h_G;
  float *d_G; // size 2 for the mus
  checkCudaErrors(hipMalloc(&d_G,  2 * sizeof(float)));
  double mu[2];
  GPU_sum<<<(Temp.n_elem-1)/256+1, 256>>>(d_G, Temp);
}
